#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstring>
#include <cmath>
#include <stdlib.h>
#include<sys/time.h>
#define BLOCKS 16
#define THREADS 16
using namespace std;

void run_sort(void);
int compSuffixes(char *suffix1, char *suffix2, int length);

//-----------------------DO NOT CHANGE NAMES, ONLY MODIFY VALUES--------------------------------------------

//Final Values that will be compared for correctness
//You may change the function prototypes and definitions, but you need to present final results in these arrays
//-----------------------------Structures for correctness check-------------------
int **SA_Final_student;
int **L_counts_student;
char *L_student;
int F_counts_student[]={0,0,0,0};
int num_value=0;
int read_count = 0;
int read_length = 0;

__global__ void bitonic_sort_step(char **dev_values, int j, int k, int num_value)
{
  unsigned int i, ixj; /* Sorting partners: i and ixj */
  i = threadIdx.x + blockDim.x * blockIdx.x;
  ixj = i^j;
  //printf("1dev="<<dev_values[0]<<endl;
  //printf("gfdgfdgdsfg\n");
  //printf("gfdgfdgdsfg\n  1dev=%s",dev_values[0]);
  /* The threads with the lowest ids sort the array. */
  if ((ixj)>i) {
    if ((i&k)==0) {
      /* Sort ascending */
      printf("1110");
      if (dev_values[i][0]>dev_values[ixj][0]) {
      printf("2222");
        /* exchange(i,ixj); */        
        char* temp;
        temp=dev_values[i];
        dev_values[i]=dev_values[ixj];
        dev_values[ixj]=temp;
      }
    }
    if ((i&k)!=0) {
      /* Sort descending */
      if (dev_values[i][0]<dev_values[ixj][0]) {
        /* exchange(i,ixj); */
        char* temp;
        temp=dev_values[i];
        dev_values[i]=dev_values[ixj];
        dev_values[ixj]=temp;
      }
    }
  }
}
void bitonic_sort(char **values)
{
  char **dev_values;
  size_t size = num_value * sizeof(char);

  hipMalloc((void***) &(&dev_values), read_count);  
  for(int i=0;i<read_count;i++)
    hipMalloc((void**) &(dev_values[i]), size);
  hipMemcpy(dev_values, values, size, hipMemcpyHostToDevice);

  dim3 blocks(BLOCKS,1);    /* Number of blocks   */
  dim3 threads(THREADS,1);  /* Number of threads  */

  int j, k;
  /* Major step */
  for (k = 2; k <= num_value; k <<= 1) {
    /* Minor step */
    for (j=k>>1; j>0; j=j>>1) {
      bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k, num_value);
    }
  }
  for(int i=0;i<sizeof(values);i++)
    cout<<"values="<<values[i]<<endl;
  cout<<"========================="<<endl;
  hipMemcpy(values, dev_values, size, hipMemcpyDeviceToHost);
  //printf("1dev=%s",dev_values);
  //cout<<"dev="<<dev_values<<endl;
  //for(int i=0; i<num_value; ++i){
      //for(int j=0; j<num_value; ++j)
          //cout<<dev_values[j][i];
      //cout<<endl;
 // }
  for(int i=0;i<sizeof(values);i++)
    cout<<"values="<<values[i]<<endl;
  /*for(int i=0;i<sizeof(dev_values);i++){
    cout<<"dev="<<dev_values[i]<<endl;
  }*/
  //cout<<"dev="<<dev_values[0]<<endl;
  hipFree(dev_values);
}


//Calculates the final FM-Index
int** makeFMIndex_student(char ***suffixes, int read_count, int read_length, int F_count[], char *L_student){
    int i, j;

    SA_Final_student=(int**)malloc(read_count*read_length*sizeof(int*));
    for(i=0;i<read_count*read_length;i++)
        SA_Final_student[i]=(int*)malloc(2*sizeof(int));

    //Temporary storage for collecting together all suffixes
    char **temp_suffixes=(char**)malloc(read_count*read_length*sizeof(char*));

    //Initalization of temporary storage
    for(i=0;i<read_count;i++){
        for(j=0;j<read_length;j++){
            temp_suffixes[i*read_length+j]=(char*)malloc(num_value*sizeof(char));
            memcpy(&temp_suffixes[i*read_length+j], &suffixes[i][j],read_length*sizeof(char));
            SA_Final_student[i*read_length+j][0]=j;
            SA_Final_student[i*read_length+j][1]=i;
        }
    }
    
    char *temp=(char*)malloc(read_length*sizeof(char));
    
    int **L_count=(int**)malloc(read_length*read_count*sizeof(int*));
    for(i=0;i<read_length*read_count;i++){
        L_count[i]=(int*)malloc(4*sizeof(int));
        for(j=0;j<4;j++){
            L_count[i][j]=0;
        }
    }

    //run_sort();
    //Focus on improving this for evaluation purpose
    //Sorting of suffixes
    /*for(i=0;i<read_count*read_length-1;i++){
        for(j=0;j<read_count*read_length-i-1;j++){
            if(compSuffixes(temp_suffixes[j], temp_suffixes[j+1], read_length)>0){
                memcpy(temp, temp_suffixes[j], read_length*sizeof(char));
                memcpy(temp_suffixes[j], temp_suffixes[j+1], read_length*sizeof(char));
                memcpy(temp_suffixes[j+1], temp, read_length*sizeof(char));
                int temp_int = SA_Final_student[j][0];
                SA_Final_student[j][0]=SA_Final_student[j+1][0];
                SA_Final_student[j+1][0]=temp_int;
                temp_int = SA_Final_student[j][1];
                SA_Final_student[j][1]=SA_Final_student[j+1][1];
                SA_Final_student[j+1][1]=temp_int;
            }
        }
    }*/
	bitonic_sort(temp_suffixes);
    free(temp);
    char this_F = '$';
    j=0;
    
    //Calculation of F_count's
    for(i=0;i<read_count*read_length;i++){
        int count=0;
        while(temp_suffixes[i][0]==this_F){
            count++;i++;
        }
        F_count[j++]=j==0?count:count+1;
        this_F = temp_suffixes[i][0];
        if(temp_suffixes[i][0]=='T')
            break;
    }
    
    //Calculation of L_student's and L_count's
    for(i=0;i<read_count*read_length;i++){
        char ch = temp_suffixes[i][read_length-1];
        L_student[i]=ch;
        if(i>0){
            for(int k=0;k<4;k++)
                L_count[i][k]=L_count[i-1][k];
        }
        if(ch=='A')
            L_count[i][0]++;
        else if(ch=='C')
            L_count[i][1]++;
        else if(ch=='G')
            L_count[i][2]++;
        else if(ch=='T')
            L_count[i][3]++;
    }
    return L_count;
}
//--------------------------------------------------------------------------------

//----------------------------------------------------------------------------------------------------------


//-----------------------DO NOT CHANGE--------------------------------------------

//int read_count = 0;
//int read_length = 0;

int **SA_Final;
int **L_counts;
char *L;
int F_counts[]={0,0,0,0};


//Read file to get reads
char** inputReads(char *file_path, int *read_count, int *length){
    FILE *read_file = fopen(file_path, "r");
    int ch, lines=0;
    char **reads;
    do                                                                                                 
    {                                                                                                  
        ch = fgetc(read_file);                                                                            
        if (ch == '\n')                                                                                
            lines++;                                                                                   
    } while (ch != EOF);
    rewind(read_file);
    reads=(char**)malloc(lines*sizeof(char*));
    *read_count = lines;
    int i = 0;                                                                                         
    size_t len = 0;                                                                                    
    for(i = 0; i < lines; i++)                                                                         
    {
        reads[i] = NULL;
        len = 0;                                                                                
        getline(&reads[i], &len, read_file);
    }                                                                                                  
    fclose(read_file);
    int j=0;
    while(reads[0][j]!='\n')
        j++;
    *length = j+1;
    for(i=0;i<lines;i++)
        reads[i][j]='$';
	int temp = (int)log2(*length);
	num_value = pow(2,temp);
    return reads;
}
//Check correctness of values
int checker(){
    int correct = 1;
    for(int i=0; i<read_count*read_length;i++){
        if(L_student[i]!=L[i]){
            //cout<<"L_student[i]!=L[i]"<<endl;
            correct = 0;
        }
            
        for(int j=0;j<2;j++){
            if(SA_Final_student[i][j]!=SA_Final[i][j]){
                //cout<<"SA_Final_student[i][j]!=SA_Final[i][j]"<<endl;
                //cout<<SA_Final_student[i][j]<<" "<<SA_Final[i][j]<<endl;
                correct = 0;
            }
                
        }
        for(int j=0;j<4;j++){
            if(L_counts_student[i][j]!=L_counts[i][j]){
                //cout<<"L_counts_student[i][j]!=L_counts[i][j]"<<endl;
                correct = 0;
            }
                
        }
    }
    for(int i=0;i<4;i++){
        if(F_counts_student[i]!=F_counts[i]){
            //cout<<"F_counts_student[i]!=F_counts[i]"<<endl;
            correct = 0;
        }
           
    }
    return correct;
}

//Rotate read by 1 character
void rotateRead(char *read, char *rotatedRead, int length){
    for(int i=0;i<length-1;i++)
        rotatedRead[i]=read[i+1];
    rotatedRead[length-1]=read[0];
}

//Generate Sufixes and their SA's for a read
char** generateSuffixes(char *read, int length, int read_id){
    char **suffixes=(char**)malloc(length*sizeof(char*));
    suffixes[0]=(char*)malloc(length*sizeof(char));
    for(int j=0;j<length;j++)
        suffixes[0][j]=read[j];
    for(int i=1;i<length;i++){
        suffixes[i]=(char*)malloc(length*sizeof(char));
        rotateRead(suffixes[i-1], suffixes[i], length);
    }
    return suffixes;
}

//Comparator for Suffixes
int compSuffixes(char *suffix1, char *suffix2, int length){
    int ret = 0;
    for(int i=0;i<length;i++){
        if(suffix1[i]>suffix2[i])
            return 1;
        else if(suffix1[i]<suffix2[i])
            return -1;
    }
    return ret;
}


//Calculates the final FM-Index
int** makeFMIndex(char ***suffixes, int read_count, int read_length, int F_count[], char *L){
    int i, j;

    SA_Final=(int**)malloc(read_count*read_length*sizeof(int*));
    for(i=0;i<read_count*read_length;i++)
        SA_Final[i]=(int*)malloc(2*sizeof(int));

    //Temporary storage for collecting together all suffixes
    char **temp_suffixes=(char**)malloc(read_count*read_length*sizeof(char*));

    //Initalization of temporary storage
    for(i=0;i<read_count;i++){
        for(j=0;j<read_length;j++){
            temp_suffixes[i*read_length+j]=(char*)malloc(read_length*sizeof(char));
            memcpy(&temp_suffixes[i*read_length+j], &suffixes[i][j],read_length*sizeof(char));
            SA_Final[i*read_length+j][0]=j;
            SA_Final[i*read_length+j][1]=i;
        }
    }
    
    char *temp=(char*)malloc(read_length*sizeof(char));
    
    int **L_count=(int**)malloc(read_length*read_count*sizeof(int*));
    for(i=0;i<read_length*read_count;i++){
        L_count[i]=(int*)malloc(4*sizeof(int));
        for(j=0;j<4;j++){
            L_count[i][j]=0;
        }
    }

    
    //Focus on improving this for evaluation purpose
    //Sorting of suffixes
    for(i=0;i<read_count*read_length-1;i++){
        for(j=0;j<read_count*read_length-i-1;j++){
            if(compSuffixes(temp_suffixes[j], temp_suffixes[j+1], read_length)>0){
                memcpy(temp, temp_suffixes[j], read_length*sizeof(char));
                memcpy(temp_suffixes[j], temp_suffixes[j+1], read_length*sizeof(char));
                memcpy(temp_suffixes[j+1], temp, read_length*sizeof(char));
                int temp_int = SA_Final[j][0];
                SA_Final[j][0]=SA_Final[j+1][0];
                SA_Final[j+1][0]=temp_int;
                temp_int = SA_Final[j][1];
                SA_Final[j][1]=SA_Final[j+1][1];
                SA_Final[j+1][1]=temp_int;
            }
        }
    }

    free(temp);
    char this_F = '$';
    j=0;
    
    //Calculation of F_count's
    for(i=0;i<read_count*read_length;i++){
        int count=0;
        while(temp_suffixes[i][0]==this_F){
            count++;i++;
        }
        F_count[j++]=j==0?count:count+1;
        this_F = temp_suffixes[i][0];
        if(temp_suffixes[i][0]=='T')
            break;
    }
    
    //Calculation of L's and L_count's
    for(i=0;i<read_count*read_length;i++){
        char ch = temp_suffixes[i][read_length-1];
        L[i]=ch;
        if(i>0){
            for(int k=0;k<4;k++)
                L_count[i][k]=L_count[i-1][k];
        }
        if(ch=='A')
            L_count[i][0]++;
        else if(ch=='C')
            L_count[i][1]++;
        else if(ch=='G')
            L_count[i][2]++;
        else if(ch=='T')
            L_count[i][3]++;
    }

    return L_count;
}

//-----------------------DO NOT CHANGE--------------------------------------------

int main(int argc, char *argv[]){

    char **reads = inputReads(argv[1], &read_count, &read_length);//Input reads from file
    char ***suffixes=(char***)malloc(read_count*sizeof(char**));//Storage for read-wise suffixes
        
    //-----------------------------Structures for correctness check----------------------------------------------
    L=(char*)malloc(read_count*read_length*sizeof(char*));//Final storage for last column of sorted suffixes
    L_student=(char*)malloc(read_count*read_length*sizeof(char*));//Final storage for last column of sorted suffixes
    //-----------------------------Structures for correctness check----------------------------------------------
    
    //-----------Default implementation----------------
    //-----------Time capture start--------------------
    struct timeval  TimeValue_Start;
    struct timeval  TimeValue_Final;
    struct timezone TimeZone_Start;
    struct timezone TimeZone_Final;
    long time_start, time_end;
    double time_overhead_default, time_overhead_student;

    gettimeofday(&TimeValue_Start, &TimeZone_Start);

    //Generate read-wise suffixes
    for(int i=0;i<read_count;i++){
        suffixes[i]=generateSuffixes(reads[i], read_length, i);
    }
    
    //Calculate finl FM-Index
    L_counts = makeFMIndex(suffixes, read_count, read_length, F_counts, L);
    
    gettimeofday(&TimeValue_Final, &TimeZone_Final);
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead_default = (time_end - time_start)/1000000.0;
    //------------Time capture end----------------------
    //--------------------------------------------------

    //-----------Your implementations------------------
    gettimeofday(&TimeValue_Start, &TimeZone_Start);
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    //-----------Call your functions here--------------------

    //Generate read-wise suffixes
    for(int i=0;i<read_count;i++){
        suffixes[i]=generateSuffixes(reads[i], read_length, i);
    }

    //Calculate finl FM-Index
    L_counts_student = makeFMIndex_student(suffixes, read_count, read_length, F_counts_student, L_student);

    //-----------Call your functions here--------------------
    gettimeofday(&TimeValue_Final, &TimeZone_Final);
    time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead_student = (time_end - time_start)/1000000.0;
    //--------------------------------------------------

 
    //----------------For debug purpose only-----------------
    //for(int i=0;i<read_count*read_length;i++)        
    //    cout<<L[i]<<"\t"<<SA_Final[i][0]<<","<<SA_Final[i][1]<<"\t"<<L_counts[i][0]<<","<<L_counts[i][1]<<","<<L_counts[i][2]<<","<<L_counts[i][3]<<endl;
    //--------------------------------------------------

    //---------------Correction check and speedup calculation----------------------
    float speedup=0.0;
    if(checker()==1)
        speedup = time_overhead_default/time_overhead_student;
    else
        cout<<"X"<<endl;
    cout<<"time_overhead_default="<<time_overhead_default<<endl;
    cout<<"time_overhead_student="<<time_overhead_student<<endl;
    cout<<"Speedup="<<speedup<<endl;
    //-----------------------------------------------------------------------------
    return 0;
}
