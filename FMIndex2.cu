
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstring>
#include <cmath>
#include <stdlib.h>
#include<sys/time.h>
#define BLOCKS 1
#define THREADS 256
using namespace std;

void run_sort(void);
int compSuffixes(char *suffix1, char *suffix2, int length);

//-----------------------DO NOT CHANGE NAMES, ONLY MODIFY VALUES--------------------------------------------

//Final Values that will be compared for correctness
//You may change the function prototypes and definitions, but you need to present final results in these arrays
//-----------------------------Structures for correctness check-------------------
int **SA_Final_student;
int **L_counts_student;
char *L_student;
int F_counts_student[]={0,0,0,0};
int num_value=0;
int read_count = 0;
int read_length = 0;

__global__ void bitonic_sort_step(char *dev_values, char* result, int j, int k, int num_value, int read_length){
    //printf("gfdgfdgdsfg\n");
    int flag = 0;
	unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i^j;
    
    //printf("enter GPU2\n");
    //printf("sizeof_values = %d\n",  );
    //printf("original string=");
    //for(int k=0;k<num_value*num_value;k++)
        //printf("%c", dev_values[k]);
    //if(i==1) printf("original string:%s\n", dev_values[0]);
    //if(i==2)
        //for(int kk=0; kk<num_value*read_length; k = k+1) 
            //printf("%c", dev_values[kk]);
    //printf("original string=");
    /*for(int kk=0;kk<num_value;kk++){
        
        //dev_values[(kk+1)*read_length-1]='\0';
        if((kk==0 || kk==1||kk==2||kk==3||kk==4)&& i==1){
            printf("%d\tdev_values=%s\n",k,&dev_values[kk*read_length]);
            //printf("========== i=%d ==========\n",k);
        }
    }*/
    /* The threads with the lowest ids sort the array. */
    flag = 0;
    int fir, sec;
    fir = 0;
    sec = 32;
    //printf("%d and %d\n", i, ixj);
    if(i==0)printf("%c and %c\n", dev_values[fir*65], dev_values[sec*65]);
    for(int l=0;l<read_length;l++){
        //if(i*read_length+l==0 && ixj*read_length+l==64*65)printf("%d, %d, %c and %c\n", i*read_length+l, ixj*read_length+l, dev_values[i*read_length+l], dev_values[ixj*read_length+l]);
        if(dev_values[i*read_length+l]>dev_values[ixj*read_length+l]){
            if(i==0&&i*read_length+l==fir*65 && ixj*read_length+l==sec*65)printf(">>>>>>>>>>>>>>>>>>\n");
            flag = 1;
            break;
        }
        else if(dev_values[i*read_length+l]<dev_values[ixj*read_length+l]){
            if(i==0&&i*read_length+l==fir*65 && ixj*read_length+l==sec*65)printf("<<<<<<<<<<<<<<<<<<<<\n");
            flag = -1;
            break;
        }
        if(i==0&&i*read_length+l==fir*65 && ixj*read_length+l==sec*65)printf("=========================\n");
        flag = 0;
    }
    if ((ixj)>i) {
        if ((i&k)==0) {
            // Sort ascending //
            //printf("1110");
            //for(int m=0;m<num_value;m++){

                if (flag==1) {
                    //printf("2222");
                    // exchange(i,ixj); //      
                    char* temp = new char[read_length];
                    for(int kk=0;kk<read_length;kk++){
                        temp[kk] = dev_values[i*read_length+kk];
                        dev_values[i*read_length+kk] = dev_values[ixj*read_length+kk];
                        dev_values[ixj*read_length+kk] = temp[kk];
                    }
                    //break;
                }
            //} 
        }
        if ((i&k)!=0) {
            // Sort descending 
            for(int m=0;m<num_value;m++){
                if (flag==-1) {
                    //printf("2222");
                    // exchange(i,ixj);         
                    char* temp = new char[read_length];
                    for(int kk=0;kk<read_length;kk++){
                        temp[kk] = dev_values[i*read_length+kk];
                        dev_values[i*read_length+kk] = dev_values[ixj*read_length+kk];
                        dev_values[ixj*read_length+kk] = temp[kk];
                    }
                }
            }
        }
    }
}
void bitonic_sort(char **values, int read_length){
    char *dev_values;
    size_t size = num_value * sizeof(char);
    char *temp;
    char* temp_char = new char[read_length];
    char* result;
    temp = (char*)malloc(read_length*size);
    cout<<"read_length="<<read_length<<endl;
    for(int i=0;i<read_length;i++){
        temp_char[i]='T';
    }
    cout<<"num_value="<<num_value<<endl;
    for(int i=0;i<num_value;i++){
        if(i<read_length)
            memcpy(&temp[i*read_length],values[i],read_length*sizeof(char));
        else{
            cout<<"into if "<<endl;
            memcpy(&temp[i*read_length],temp_char,read_length*sizeof(char));   
        }
    }
    cout<<"==============================="<<endl;
    cout<<"temp_char="<<temp_char<<endl;
    cout<<"==============================="<<endl;
    cout<<"temp="<<temp<<endl;
    cout<<"==============================="<<endl;
	for(int i=0;i<read_length;i++)
        cout<<"values="<<values[i]<<endl;
	cout<<"========================="<<endl;
    fprintf(stderr,"enter bitonic sort\n");
    hipMalloc((void**) &dev_values, size*read_length);
    cout<<"dfsfsdfsdf"<<size<<num_value<<endl;
    hipMalloc((void**) &result, size*num_value);
    
    
    hipMemcpy(dev_values, temp, read_length*size, hipMemcpyHostToDevice);
    fprintf(stderr,"finish pushing data into device\n");

    dim3 blocks(BLOCKS,1);    /* Number of blocks   */
    dim3 threads(THREADS,1);  /* Number of threads  */

    int j, k;
    /* Major step */
    for (k = 2; k <= num_value; k <<= 1) {
        //* Minor step */
        for (j=k>>1; j>0; j=j>>1) {
			fprintf(stderr,"before entering GPU\n");
			bitonic_sort_step<<<1, 1>>>(dev_values,result, j, k, num_value,read_length);
			fprintf(stderr,"leaving GPU\n");
        }
    }
   
		
    //for(int i=0;i<read_length;i++){
        //cudaMemcpy(values[i], &dev_values[i*read_length], read_length*sizeof(char), cudaMemcpyDeviceToHost);       
    //}
    hipMemcpy(temp, dev_values, read_length*size, hipMemcpyDeviceToHost);
    
    cout<<"begin teeeeeeeeeeeeeeeeeeeeeeeeeeeeeemp"<<endl;
    for(int i=0;i<read_length;i++){
        memcpy(values[i],&temp[i*read_length],read_length*sizeof(char));
        cout<<values[i]<<endl;
    }  
    
    cout<<"tttttttttttttttteeeeeeeeeeemmmmmmmmmmmmmmmpppppppppppppppppppppp"<<endl;
    cout<<temp<<endl;
    //for(int i=0;i<read_length;i++)
        //cout<<"i="<<i<<"values=\t\t"<<values[i]<<endl;
    hipFree(dev_values);
}


//Calculates the final FM-Index
int** makeFMIndex_student(char ***suffixes, int read_count, int read_length, int F_count[], char *L_student){
    int i, j;
    int temp_stu = ceil(log2((float)read_length));
	num_value = pow(2,temp_stu);
    cout<<"======================="<<endl;
    cout<<"read_length = "<<read_length<<endl;
    cout<<"======================="<<endl;
    cout<<"======================="<<endl;
    cout<<"temp_stu = "<<temp_stu<<endl;
    cout<<"======================="<<endl;
    cout<<"======================="<<endl;
    cout<<"num_value = "<<num_value<<endl;
    cout<<"======================="<<endl;
    
    SA_Final_student=(int**)malloc(read_count*read_length*sizeof(int*));
    for(i=0;i<read_count*read_length;i++){
        
        SA_Final_student[i]=(int*)malloc(2*sizeof(int));
    }
    //Temporary storage for collecting together all suffixes
    char **temp_suffixes=(char**)malloc(read_count*read_length*sizeof(char*));

    //Initalization of temporary storage
    for(i=0;i<read_count;i++){
        for(j=0;j<read_length;j++){
            temp_suffixes[i*read_length+j]=(char*)malloc(num_value*sizeof(char));
            memcpy(&temp_suffixes[i*read_length+j], &suffixes[i][j],read_length*sizeof(char));
            SA_Final_student[i*read_length+j][0]=j;
            SA_Final_student[i*read_length+j][1]=i;
        }
    }
    cout<<"read_count = "<<read_count<<endl;
    cout<<"read_length = "<<read_length<<endl;
    char *temp=(char*)malloc(read_length*sizeof(char));
    
    int **L_count=(int**)malloc(read_length*read_count*sizeof(int*));
    for(i=0;i<read_length*read_count;i++){
        L_count[i]=(int*)malloc(4*sizeof(int));
        for(j=0;j<4;j++){
            L_count[i][j]=0;
        }
    }

    //run_sort();
    //Focus on improving this for evaluation purpose
    //Sorting of suffixes
    /*for(i=0;i<read_count*read_length-1;i++){
        for(j=0;j<read_count*read_length-i-1;j++){
            if(compSuffixes(temp_suffixes[j], temp_suffixes[j+1], read_length)>0){
                memcpy(temp, temp_suffixes[j], read_length*sizeof(char));
                memcpy(temp_suffixes[j], temp_suffixes[j+1], read_length*sizeof(char));
                memcpy(temp_suffixes[j+1], temp, read_length*sizeof(char));
                int temp_int = SA_Final_student[j][0];
                SA_Final_student[j][0]=SA_Final_student[j+1][0];
                SA_Final_student[j+1][0]=temp_int;
                temp_int = SA_Final_student[j][1];
                SA_Final_student[j][1]=SA_Final_student[j+1][1];
                SA_Final_student[j+1][1]=temp_int;
            }
        }
    }*/
	bitonic_sort(temp_suffixes,read_length*read_count);
    free(temp);
    char this_F = '$';
    j=0;
    
    //Calculation of F_count's
    for(i=0;i<read_count*read_length;i++){
        int count=0;
        while(temp_suffixes[i][0]==this_F){
            count++;i++;
        }
        F_count[j++]=j==0?count:count+1;
        this_F = temp_suffixes[i][0];
        if(temp_suffixes[i][0]=='T')
            break;
    }
    
    //Calculation of L_student's and L_count's
    for(i=0;i<read_count*read_length;i++){
        char ch = temp_suffixes[i][read_length-1];
        L_student[i]=ch;
        if(i>0){
            for(int k=0;k<4;k++)
                L_count[i][k]=L_count[i-1][k];
        }
        if(ch=='A')
            L_count[i][0]++;
        else if(ch=='C')
            L_count[i][1]++;
        else if(ch=='G')
            L_count[i][2]++;
        else if(ch=='T')
            L_count[i][3]++;
    }
    return L_count;
}
//--------------------------------------------------------------------------------

//----------------------------------------------------------------------------------------------------------


//-----------------------DO NOT CHANGE--------------------------------------------

//int read_count = 0;
//int read_length = 0;

int **SA_Final;
int **L_counts;
char *L;
int F_counts[]={0,0,0,0};


//Read file to get reads
char** inputReads(char *file_path, int *read_count, int *length){
    FILE *read_file = fopen(file_path, "r");
    int ch, lines=0;
    char **reads;
    do                                                                                                 
    {                                                                                                  
        ch = fgetc(read_file);                                                                            
        if (ch == '\n')                                                                                
            lines++;                                                                                   
    } while (ch != EOF);
    rewind(read_file);
    reads=(char**)malloc(lines*sizeof(char*));
    *read_count = lines;
    int i = 0;                                                                                         
    size_t len = 0;                                                                                    
    for(i = 0; i < lines; i++)                                                                         
    {
        reads[i] = NULL;
        len = 0;                                                                                
        getline(&reads[i], &len, read_file);
    }                                                                                                  
    fclose(read_file);
    int j=0;
    while(reads[0][j]!='\n')
        j++;
    *length = j+1;
    for(i=0;i<lines;i++)
        reads[i][j]='$';
	int temp = log2((float)*length);
	num_value = pow(2,temp);
    return reads;
}
//Check correctness of values
int checker(){
    int correct = 1;
    for(int i=0; i<read_count*read_length;i++){
        if(L_student[i]!=L[i]){
            //cout<<"L_student[i]!=L[i]"<<endl;
            correct = 0;
        }
            
        for(int j=0;j<2;j++){
            if(SA_Final_student[i][j]!=SA_Final[i][j]){
                //cout<<"SA_Final_student[i][j]!=SA_Final[i][j]"<<endl;
                //cout<<SA_Final_student[i][j]<<" "<<SA_Final[i][j]<<endl;
                correct = 0;
            }
                
        }
        for(int j=0;j<4;j++){
            if(L_counts_student[i][j]!=L_counts[i][j]){
                //cout<<"L_counts_student[i][j]!=L_counts[i][j]"<<endl;
                correct = 0;
            }
                
        }
    }
    for(int i=0;i<4;i++){
        if(F_counts_student[i]!=F_counts[i]){
            //cout<<"F_counts_student[i]!=F_counts[i]"<<endl;
            correct = 0;
        }
           
    }
    return correct;
}

//Rotate read by 1 character
void rotateRead(char *read, char *rotatedRead, int length){
    for(int i=0;i<length-1;i++)
        rotatedRead[i]=read[i+1];
    rotatedRead[length-1]=read[0];
}

//Generate Sufixes and their SA's for a read
char** generateSuffixes(char *read, int length, int read_id){
    char **suffixes=(char**)malloc(length*sizeof(char*));
    suffixes[0]=(char*)malloc(length*sizeof(char));
    for(int j=0;j<length;j++)
        suffixes[0][j]=read[j];
    for(int i=1;i<length;i++){
        suffixes[i]=(char*)malloc(length*sizeof(char));
        rotateRead(suffixes[i-1], suffixes[i], length);
    }
    return suffixes;
}

//Comparator for Suffixes
int compSuffixes(char *suffix1, char *suffix2, int length){
    int ret = 0;
    for(int i=0;i<length;i++){
        if(suffix1[i]>suffix2[i])
            return 1;
        else if(suffix1[i]<suffix2[i])
            return -1;
    }
    return ret;
}


//Calculates the final FM-Index
int** makeFMIndex(char ***suffixes, int read_count, int read_length, int F_count[], char *L){
    int i, j;

    SA_Final=(int**)malloc(read_count*read_length*sizeof(int*));
    for(i=0;i<read_count*read_length;i++)
        SA_Final[i]=(int*)malloc(2*sizeof(int));

    //Temporary storage for collecting together all suffixes
    char **temp_suffixes=(char**)malloc(read_count*read_length*sizeof(char*));

    //Initalization of temporary storage
    for(i=0;i<read_count;i++){
        for(j=0;j<read_length;j++){
            temp_suffixes[i*read_length+j]=(char*)malloc(read_length*sizeof(char));
            memcpy(&temp_suffixes[i*read_length+j], &suffixes[i][j],read_length*sizeof(char));
            SA_Final[i*read_length+j][0]=j;
            SA_Final[i*read_length+j][1]=i;
        }
    }
    
    char *temp=(char*)malloc(read_length*sizeof(char));
    
    int **L_count=(int**)malloc(read_length*read_count*sizeof(int*));
    for(i=0;i<read_length*read_count;i++){
        L_count[i]=(int*)malloc(4*sizeof(int));
        for(j=0;j<4;j++){
            L_count[i][j]=0;
        }
    }

    
    //Focus on improving this for evaluation purpose
    //Sorting of suffixes
    for(i=0;i<read_count*read_length-1;i++){
        for(j=0;j<read_count*read_length-i-1;j++){
            if(compSuffixes(temp_suffixes[j], temp_suffixes[j+1], read_length)>0){
                memcpy(temp, temp_suffixes[j], read_length*sizeof(char));
                memcpy(temp_suffixes[j], temp_suffixes[j+1], read_length*sizeof(char));
                memcpy(temp_suffixes[j+1], temp, read_length*sizeof(char));
                int temp_int = SA_Final[j][0];
                SA_Final[j][0]=SA_Final[j+1][0];
                SA_Final[j+1][0]=temp_int;
                temp_int = SA_Final[j][1];
                SA_Final[j][1]=SA_Final[j+1][1];
                SA_Final[j+1][1]=temp_int;
            }
        }
    }

    free(temp);
    char this_F = '$';
    j=0;
    
    //Calculation of F_count's
    for(i=0;i<read_count*read_length;i++){
        int count=0;
        while(temp_suffixes[i][0]==this_F){
            count++;i++;
        }
        F_count[j++]=j==0?count:count+1;
        this_F = temp_suffixes[i][0];
        if(temp_suffixes[i][0]=='T')
            break;
    }
    
    //Calculation of L's and L_count's
    for(i=0;i<read_count*read_length;i++){
        char ch = temp_suffixes[i][read_length-1];
        L[i]=ch;
        if(i>0){
            for(int k=0;k<4;k++)
                L_count[i][k]=L_count[i-1][k];
        }
        if(ch=='A')
            L_count[i][0]++;
        else if(ch=='C')
            L_count[i][1]++;
        else if(ch=='G')
            L_count[i][2]++;
        else if(ch=='T')
            L_count[i][3]++;
    }

    return L_count;
}

//-----------------------DO NOT CHANGE--------------------------------------------

int main(int argc, char *argv[]){

    char **reads = inputReads(argv[1], &read_count, &read_length);//Input reads from file
    char ***suffixes=(char***)malloc(read_count*sizeof(char**));//Storage for read-wise suffixes
        
    //-----------------------------Structures for correctness check----------------------------------------------
    L=(char*)malloc(read_count*read_length*sizeof(char*));//Final storage for last column of sorted suffixes
    L_student=(char*)malloc(read_count*read_length*sizeof(char*));//Final storage for last column of sorted suffixes
    //-----------------------------Structures for correctness check----------------------------------------------
    
    //-----------Default implementation----------------
    //-----------Time capture start--------------------
    struct timeval  TimeValue_Start;
    struct timeval  TimeValue_Final;
    struct timezone TimeZone_Start;
    struct timezone TimeZone_Final;
    long time_start, time_end;
    double time_overhead_default, time_overhead_student;

    gettimeofday(&TimeValue_Start, &TimeZone_Start);

    //Generate read-wise suffixes
    for(int i=0;i<read_count;i++){
        suffixes[i]=generateSuffixes(reads[i], read_length, i);
    }
    
    //Calculate finl FM-Index
    L_counts = makeFMIndex(suffixes, read_count, read_length, F_counts, L);
    
    gettimeofday(&TimeValue_Final, &TimeZone_Final);
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead_default = (time_end - time_start)/1000000.0;
    //------------Time capture end----------------------
    //--------------------------------------------------

    //-----------Your implementations------------------
    gettimeofday(&TimeValue_Start, &TimeZone_Start);
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    //-----------Call your functions here--------------------

    //Generate read-wise suffixes
    for(int i=0;i<read_count;i++){
        suffixes[i]=generateSuffixes(reads[i], read_length, i);
    }

    //Calculate finl FM-Index
    L_counts_student = makeFMIndex_student(suffixes, read_count, read_length, F_counts_student, L_student);

    //-----------Call your functions here--------------------
    gettimeofday(&TimeValue_Final, &TimeZone_Final);
    time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead_student = (time_end - time_start)/1000000.0;
    //--------------------------------------------------

 
    //----------------For debug purpose only-----------------
    //for(int i=0;i<read_count*read_length;i++)        
    //    cout<<L[i]<<"\t"<<SA_Final[i][0]<<","<<SA_Final[i][1]<<"\t"<<L_counts[i][0]<<","<<L_counts[i][1]<<","<<L_counts[i][2]<<","<<L_counts[i][3]<<endl;
    //--------------------------------------------------

    //---------------Correction check and speedup calculation----------------------
    float speedup=0.0;
    if(checker()==1)
        speedup = time_overhead_default/time_overhead_student;
    else
        cout<<"X"<<endl;
    cout<<"time_overhead_default="<<time_overhead_default<<endl;
    cout<<"time_overhead_student="<<time_overhead_student<<endl;
    cout<<"Speedup="<<speedup<<endl;
    //-----------------------------------------------------------------------------
    return 0;
}
