#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <iomanip>
#include <cstring>
#include <cmath>
#include <stdlib.h>
#include<sys/time.h>
#define MAX 10000000
using namespace std;
int **L_counts;
void run_sort(void);
int compSuffixes(char *suffix1, char *suffix2, int length);

//-----------------------DO NOT CHANGE NAMES, ONLY MODIFY VALUES--------------------------------------------

//Final Values that will be compared for correctness
//You may change the function prototypes and definitions, but you need to present final results in these arrays
//-----------------------------Structures for correctness check-------------------
int **SA_Final_student;
int **L_counts_student;
char *L_student;
int F_counts_student[]={0,0,0,0};
int num_value = 0;
int read_count = 0;
int read_length = 0;
int BLOCKS, THREADS;

__global__ void bitonic_sort_step(char *dev_values, int j, int k, int num_value, int read_length, int read_count, int *dev_SA_Final_1, int *dev_SA_Final_2){
    //printf("gfdgfdgdsfg\n");
    int flag = 0;
	unsigned int i, ixj; /* Sorting partners: i and ixj */
    i = threadIdx.x + blockDim.x * blockIdx.x;
    ixj = i^j;
    

    /* The threads with the lowest ids sort the array. */
    flag = 0;

    if ((ixj)>i) {
        for(int l=0;l<read_length;l++){
            if(dev_values[i*read_length+l]>dev_values[ixj*read_length+l]){
                //if(i==0&&i*read_length+l==fir*65 && ixj*read_length+l==sec*65)printf(">>>>>>>>>>>>>>>>>>\n");
                flag = 1;
                break;
            }
            else if(dev_values[i*read_length+l]<dev_values[ixj*read_length+l]){
                //if(i==0&&i*read_length+l==fir*65 && ixj*read_length+l==sec*65)printf("<<<<<<<<<<<<<<<<<<<<\n");
                flag = -1;
                break;
            }
            //if(i==0&&i*read_length+l==fir*65 && ixj*read_length+l==sec*65)printf("=========================\n");
            flag = 0;
            
        }
        //printf("i=%d, ixj=%d, sorting result flag = %d\n",i,ixj,flag);
        

        if ((i&k)==0) {
            // Sort ascending //
            //printf("1110");
            //for(int m=0;m<num_value;m++){

                if (flag==1) {
                    //printf("3333, %d, %d\n", i, ixj);  
                    char* temp;
					temp = (char*)malloc(sizeof(char)*read_length);
					memcpy(temp, &dev_values[i*read_length], read_length*sizeof(char));
					memcpy(&dev_values[i*read_length], &dev_values[ixj*read_length], read_length*sizeof(char));
					memcpy(&dev_values[ixj*read_length], temp, read_length*sizeof(char));
                    int temp_SA_1;
                    temp_SA_1 = dev_SA_Final_1[i];
                    dev_SA_Final_1[i] = dev_SA_Final_1[ixj];
                    dev_SA_Final_1[ixj] = temp_SA_1;
                    int temp_SA_2;
                    temp_SA_2 = dev_SA_Final_2[i];
                    dev_SA_Final_2[i] = dev_SA_Final_2[ixj];
                    dev_SA_Final_2[ixj] = temp_SA_2;
					free(temp);
                }
        }
        if ((i&k)!=0) {
            // Sort descending 
            
            if (flag==-1) {
                //printf("2222, %d, %d\n", i, ixj);         
                char* temp;
                temp = (char*)malloc(sizeof(char)*read_length);
                memcpy(temp, &dev_values[i*read_length], read_length*sizeof(char));
                memcpy(&dev_values[i*read_length], &dev_values[ixj*read_length], read_length*sizeof(char));
                memcpy(&dev_values[ixj*read_length], temp, read_length*sizeof(char));
                int temp_SA_1;
                temp_SA_1 = dev_SA_Final_1[i];
                dev_SA_Final_1[i] = dev_SA_Final_1[ixj];
                dev_SA_Final_1[ixj] = temp_SA_1;
                int temp_SA_2;
                temp_SA_2 = dev_SA_Final_2[i];
                dev_SA_Final_2[i] = dev_SA_Final_2[ixj];
                dev_SA_Final_2[ixj] = temp_SA_2;
				free(temp);
            }
        }
    }
}
void bitonic_sort(char **values){
    char *dev_values;
    size_t size = read_length * sizeof(char);
    char *temp;
    char *temp_char = new char[read_length];
    int* SA_Final_1 = new int[num_value];
    int* SA_Final_2 = new int[num_value];
    int* dev_SA_Final_1;
    int* dev_SA_Final_2;
    temp = (char*)malloc(num_value*size);
    for(int i=0;i<read_length;i++){
        temp_char[i]='T';
    }
    for(int i=0;i<num_value;i++){
        if(i<read_length*read_count){
            memcpy(&temp[i*read_length],values[i],read_length*sizeof(char));
            SA_Final_1[i] = SA_Final_student[i][0];
            SA_Final_2[i] = SA_Final_student[i][1];
        }
        else{
            memcpy(&temp[i*read_length],temp_char,read_length*sizeof(char));   
        }
    }
    hipMalloc((void**) &dev_values, size*num_value);
    hipMalloc((void**) &dev_SA_Final_1, num_value*sizeof(int));
    hipMalloc((void**) &dev_SA_Final_2, num_value*sizeof(int));
    
    
    hipMemcpy(dev_values, temp, num_value*size, hipMemcpyHostToDevice);
    hipMemcpy(dev_SA_Final_1, SA_Final_1, num_value*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_SA_Final_2, SA_Final_2, num_value*sizeof(int), hipMemcpyHostToDevice);

    dim3 blocks(BLOCKS,1);    /* Number of blocks   */
    dim3 threads(THREADS,1);  /* Number of threads  */

    int j, k;
    /* Major step */
    
    for (k = 2; k <= num_value; k <<= 1) {
        //* Minor step */
        for (j=k>>1; j>0; j=j>>1) {
			bitonic_sort_step<<<blocks, threads>>>(dev_values, j, k, num_value,read_length, read_count ,dev_SA_Final_1,dev_SA_Final_2);
		}
    }
      
		
    //for(int i=0;i<read_length;i++){
        //hipMemcpy(values[i], &dev_values[i*read_length], read_length*sizeof(char), hipMemcpyDeviceToHost);       
    //}
    hipMemcpy(temp, dev_values, read_length*read_count*size, hipMemcpyDeviceToHost);
    hipMemcpy(SA_Final_1, dev_SA_Final_1, num_value*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(SA_Final_2, dev_SA_Final_2, num_value*sizeof(int), hipMemcpyDeviceToHost);
   
    //cout<<"begin teeeeeeeeeeeeeeeeeeeeeeeeeeeeeemp"<<endl;
    for(int i=0;i<read_length*read_count;i++){

        memcpy(values[i],&temp[i*read_length],read_length*sizeof(char));
        SA_Final_student[i][0] = SA_Final_1[i];
        SA_Final_student[i][1] = SA_Final_2[i];
        /*if(i==0){
              for(int i=0; i<read_count*read_length;i++){
                for(int j=0;j<4;j++)
                        //cout<<"original answer for"<<i<<","<<j<<" is "<<L_counts[i][j]<<endl; 
                          printf("original answer for%d,%d is %d\n",i, j, L_counts[i][j]);
                
            }  
        } */
        //hipMemcpy(SA_Final_student[i][0], dev_SA_Final_1, read_length*size, hipMemcpyDeviceToHost);
        //cout<<values[i]<<endl;
    }  
      
    //cout<<"tttttttttttttttteeeeeeeeeeemmmmmmmmmmmmmmmpppppppppppppppppppppp"<<endl;
    //cout<<temp<<endl;
    //for(int i=0;i<read_length;i++)
        //cout<<"i="<<i<<"values=\t\t"<<values[i]<<endl;
	free(temp);
    hipFree(dev_values);
	hipFree(SA_Final_1);
	hipFree(SA_Final_2);
}


//Calculates the final FM-Index
int** makeFMIndex_student(char ***suffixes, int read_count, int read_length, int F_count[], char *L_student){

     
    int i, j;
    int temp_stu = ceil(log2((float)read_length*read_count));
	num_value = pow(2,temp_stu);
	if(num_value<=256){
		THREADS = num_value;
		BLOCKS = 1;
	}
	else{
		THREADS = 256;
		BLOCKS = num_value/THREADS;
	}
    /*cout<<"======================="<<endl;
    cout<<"read_length = "<<read_length<<endl;
    cout<<"======================="<<endl;
    cout<<"======================="<<endl;
    cout<<"temp_stu = "<<temp_stu<<endl;
    cout<<"======================="<<endl;
    cout<<"======================="<<endl;
    cout<<"num_value = "<<num_value<<endl;
    cout<<"======================="<<endl;*/
    
    SA_Final_student=(int**)malloc(read_count*read_length*sizeof(int*));

    
    for(i=0;i<read_count*read_length;i++){
        
        SA_Final_student[i]=(int*)malloc(2*sizeof(int));
    }

    //Temporary storage for collecting together all suffixes
    char **temp_suffixes=(char**)malloc(read_count*read_length*sizeof(char*));

    //Initalization of temporary storage
    for(i=0;i<read_count;i++){
        for(j=0;j<read_length;j++){
            temp_suffixes[i*read_length+j]=(char*)malloc(num_value*sizeof(char));
            memcpy(&temp_suffixes[i*read_length+j], &suffixes[i][j],read_length*sizeof(char));
            SA_Final_student[i*read_length+j][0]=j;
            SA_Final_student[i*read_length+j][1]=i;
        }
    }
    /*for(int i=0;i<read_count;i++){
        for(int j=0;j<read_length;j++)
            cout<<temp_suffixes[i*read_length+j]<<endl;
    }*/
    cout<<endl;
    cout<<"read_count = "<<read_count<<endl;
    cout<<"read_length = "<<read_length<<endl;
    char *temp=(char*)malloc(read_length*sizeof(char));
    
    int **L_count=(int**)malloc(read_length*read_count*sizeof(int*));
    for(int i=0;i<read_length*read_count;i++){
        //L_count[i]=(int*)malloc(4*sizeof(int));
		L_count[i]=new int[4];
        for(int j=0;j<4;j++){
            L_count[i][j]=0;
        }
    }

	bitonic_sort(temp_suffixes);
    
    for(int i=0;i<read_count;i++){
        for(int j=0;j<read_length;j++)
            cout<<temp_suffixes[i*read_length+j]<<endl;
    }
    free(temp);
    char this_F = '$';
    j=0;    
    //Calculation of F_count's
    for(int i=0;i<read_count*read_length;i++){
		int count=0;
        while(temp_suffixes[i][0]==this_F){
            count++;i++;
        }
        F_count[j++]=j==0?count:count+1;
        this_F = temp_suffixes[i][0];
        if(temp_suffixes[i][0]=='T')
            break;
    }
	
    //Calculation of L_student's and L_count's
    for(i=0;i<read_count*read_length;i++){
        char ch = temp_suffixes[i][read_length-1];
        L_student[i]=ch;
        if(i>0){
            for(int k=0;k<4;k++)
                L_count[i][k]=L_count[i-1][k];
        }
        if(ch=='A')
            L_count[i][0]++;
        else if(ch=='C')
            L_count[i][1]++;
        else if(ch=='G')
            L_count[i][2]++;
        else if(ch=='T')
            L_count[i][3]++;
    }
    return L_count;
}
//--------------------------------------------------------------------------------

//----------------------------------------------------------------------------------------------------------


//-----------------------DO NOT CHANGE--------------------------------------------

//int read_count = 0;
//int read_length = 0;

int **SA_Final;

char *L;
int F_counts[]={0,0,0,0};


//Read file to get reads
char** inputReads(char *file_path, int *read_count, int *length){
    FILE *read_file = fopen(file_path, "r");
    int ch, lines=0;
    char **reads;
    do                                                                                                 
    {                                                                                                  
        ch = fgetc(read_file);                                                                            
        if (ch == '\n')                                                                                
            lines++;                                                                                   
    } while (ch != EOF);
    rewind(read_file);
    reads=(char**)malloc(lines*sizeof(char*));
    *read_count = lines;
    int i = 0;                                                                                         
    size_t len = 0;                                                                                    
    for(i = 0; i < lines; i++)                                                                         
    {
        reads[i] = NULL;
        len = 0;                                                                                
        getline(&reads[i], &len, read_file);
    }                                                                                                  
    fclose(read_file);
    int j=0;
    while(reads[0][j]!='\n')
        j++;
    *length = j+1;
    for(i=0;i<lines;i++)
        reads[i][j]='$';
    return reads;
}
//Check correctness of values
int checker(){
    int correct = 1;
    cout<<"cccccccccccccchecker!!!!!!!!!!"<<endl;  
    for(int i=0; i<read_count*read_length;i++){
        
        if(L_student[i]!=L[i]){
            cout<<"L_student[i]!=L[i]"<<endl;
            correct = 0;
            cout<<"answer is "<<L_student[i]<<endl;
        }
            
        for(int j=0;j<2;j++){
            if(SA_Final_student[i][j]!=SA_Final[i][j]){
                cout<<"SA_Final_student[i][j]!=SA_Final[i][j]"<<endl;
                cout<<SA_Final_student[i][j]<<" "<<SA_Final[i][j]<<endl;
                correct = 0;  
            }               
        }
        for(int j=0;j<4;j++){
            //cout<<L_counts_student[i][j]<<endl;
            if(L_counts_student[i][j]!=L_counts[i][j]){
                cout<<"answer for"<<i<<","<<j<<" is "<<L_counts[i][j]<<endl;
                correct = 0;
            }
                
        }
    }
    for(int i=0;i<4;i++){
        if(F_counts_student[i]!=F_counts[i]){
            //cout<<"F_counts_student[i]!=F_counts[i]"<<endl;
            correct = 0;
        }
           
    }
    return correct;
}

//Rotate read by 1 character
void rotateRead(char *read, char *rotatedRead, int length){
    for(int i=0;i<length-1;i++)
        rotatedRead[i]=read[i+1];
    rotatedRead[length-1]=read[0];
}

//Generate Sufixes and their SA's for a read
char** generateSuffixes(char *read, int length, int read_id){
    char **suffixes=(char**)malloc(length*sizeof(char*));
    suffixes[0]=(char*)malloc(length*sizeof(char));
    for(int j=0;j<length;j++)
        suffixes[0][j]=read[j];
    for(int i=1;i<length;i++){
        suffixes[i]=(char*)malloc(length*sizeof(char));
        rotateRead(suffixes[i-1], suffixes[i], length);
    }
    return suffixes;
}

//Comparator for Suffixes
int compSuffixes(char *suffix1, char *suffix2, int length){
    int ret = 0;
    for(int i=0;i<length;i++){
        if(suffix1[i]>suffix2[i])
            return 1;
        else if(suffix1[i]<suffix2[i])
            return -1;
    }
    return ret;
}


//Calculates the final FM-Index
int** makeFMIndex(char ***suffixes, int read_count, int read_length, int F_count[], char *L){
    int i, j;

    SA_Final=(int**)malloc(read_count*read_length*sizeof(int*));
    for(i=0;i<read_count*read_length;i++)
        SA_Final[i]=(int*)malloc(2*sizeof(int));

    //Temporary storage for collecting together all suffixes
    char **temp_suffixes=(char**)malloc(read_count*read_length*sizeof(char*));

    //Initalization of temporary storage
    for(i=0;i<read_count;i++){
        for(j=0;j<read_length;j++){
            temp_suffixes[i*read_length+j]=(char*)malloc(read_length*sizeof(char));
            memcpy(&temp_suffixes[i*read_length+j], &suffixes[i][j],read_length*sizeof(char));
            SA_Final[i*read_length+j][0]=j;
            SA_Final[i*read_length+j][1]=i;
        }
    }
    
    char *temp=(char*)malloc(read_length*sizeof(char));
    
    int **L_count=(int**)malloc(read_length*read_count*sizeof(int*));
    for(i=0;i<read_length*read_count;i++){
        L_count[i]=(int*)malloc(4*sizeof(int));
        for(j=0;j<4;j++){
            L_count[i][j]=0;
        }
    }

    
    //Focus on improving this for evaluation purpose
    //Sorting of suffixes
    for(i=0;i<read_count*read_length-1;i++){
        for(j=0;j<read_count*read_length-i-1;j++){
            if(compSuffixes(temp_suffixes[j], temp_suffixes[j+1], read_length)>0){
                memcpy(temp, temp_suffixes[j], read_length*sizeof(char));
                memcpy(temp_suffixes[j], temp_suffixes[j+1], read_length*sizeof(char));
                memcpy(temp_suffixes[j+1], temp, read_length*sizeof(char));
                int temp_int = SA_Final[j][0];
                SA_Final[j][0]=SA_Final[j+1][0];
                SA_Final[j+1][0]=temp_int;
                temp_int = SA_Final[j][1];
                SA_Final[j][1]=SA_Final[j+1][1];
                SA_Final[j+1][1]=temp_int;
            }
        }
    }

    free(temp);
    char this_F = '$';
    j=0;
    
    //Calculation of F_count's
    for(i=0;i<read_count*read_length;i++){
        int count=0;
        while(temp_suffixes[i][0]==this_F){
            count++;i++;
        }
        F_count[j++]=j==0?count:count+1;
        this_F = temp_suffixes[i][0];
        if(temp_suffixes[i][0]=='T')
            break;
    }
    
    //Calculation of L's and L_count's
    for(i=0;i<read_count*read_length;i++){
        char ch = temp_suffixes[i][read_length-1];
        L[i]=ch;
        if(i>0){
            for(int k=0;k<4;k++)
                L_count[i][k]=L_count[i-1][k];
        }
        if(ch=='A')
            L_count[i][0]++;
        else if(ch=='C')
            L_count[i][1]++;
        else if(ch=='G')
            L_count[i][2]++;
        else if(ch=='T')
            L_count[i][3]++;
    }

    return L_count;
}

//-----------------------DO NOT CHANGE--------------------------------------------

int main(int argc, char *argv[]){

    char **reads = inputReads(argv[1], &read_count, &read_length);//Input reads from file
    char ***suffixes=(char***)malloc(read_count*sizeof(char**));//Storage for read-wise suffixes
        
    //-----------------------------Structures for correctness check----------------------------------------------
    L=(char*)malloc(read_count*read_length*sizeof(char*));//Final storage for last column of sorted suffixes
    L_student=(char*)malloc(read_count*read_length*sizeof(char*));//Final storage for last column of sorted suffixes
    //-----------------------------Structures for correctness check----------------------------------------------
    
    //-----------Default implementation----------------
    //-----------Time capture start--------------------
    struct timeval  TimeValue_Start;
    struct timeval  TimeValue_Final;
    struct timezone TimeZone_Start;
    struct timezone TimeZone_Final;
    long time_start, time_end;
    double time_overhead_default, time_overhead_student;

    gettimeofday(&TimeValue_Start, &TimeZone_Start);

    //Generate read-wise suffixes
    for(int i=0;i<read_count;i++){
        suffixes[i]=generateSuffixes(reads[i], read_length, i);
    }
    
    //Calculate finl FM-Index
    L_counts = makeFMIndex(suffixes, read_count, read_length, F_counts, L);

    
    gettimeofday(&TimeValue_Final, &TimeZone_Final);
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead_default = (time_end - time_start)/1000000.0;
    //------------Time capture end----------------------
    //--------------------------------------------------

    //-----------Your implementations------------------
    gettimeofday(&TimeValue_Start, &TimeZone_Start);
    time_start = TimeValue_Start.tv_sec * 1000000 + TimeValue_Start.tv_usec;
    //-----------Call your functions here--------------------

    //Generate read-wise suffixes
    for(int i=0;i<read_count;i++){
        suffixes[i]=generateSuffixes(reads[i], read_length, i);
    }
        
    //Calculate finl FM-Index
    L_counts_student = makeFMIndex_student(suffixes, read_count, read_length, F_counts_student, L_student);  
    //-----------Call your functions here--------------------
    gettimeofday(&TimeValue_Final, &TimeZone_Final);
    time_end = TimeValue_Final.tv_sec * 1000000 + TimeValue_Final.tv_usec;
    time_overhead_student = (time_end - time_start)/1000000.0;
    //--------------------------------------------------

 
    //----------------For debug purpose only-----------------
    //for(int i=0;i<read_count*read_length;i++)        
    //    cout<<L[i]<<"\t"<<SA_Final[i][0]<<","<<SA_Final[i][1]<<"\t"<<L_counts[i][0]<<","<<L_counts[i][1]<<","<<L_counts[i][2]<<","<<L_counts[i][3]<<endl;
    //--------------------------------------------------

    //---------------Correction check and speedup calculation----------------------
    float speedup=0.0;
    fprintf(stderr,"num_value=%d\tBLOCKS=%d\tTHREADS=%d\n",num_value,BLOCKS,THREADS);
    if(checker()==1){
		fprintf(stderr,"congratulations!!!!\n");
        speedup = time_overhead_default/time_overhead_student;
	}
    else
        cout<<"X but Speedup="<<time_overhead_default/time_overhead_student<<endl;
    cout<<"time_overhead_default="<<time_overhead_default<<endl;
    cout<<"time_overhead_student="<<time_overhead_student<<endl;
    cout<<"Speedup="<<speedup<<endl;
	fprintf(stderr,"Speedup=%f\n",speedup);
    //-----------------------------------------------------------------------------
    return 0;
}

